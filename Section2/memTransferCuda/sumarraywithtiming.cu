#include "hip/hip_runtime.h"
#include "common.h"
#include "cuda_common.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <cstring>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void sum_array_gpu(int *a, int *b, int *c, int size) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    c[gid] = a[gid] + b[gid];
  }
}

void sum_array_cpu(int *a, int *b, int *c, int size) {
  for (int i = 0; i < size; i++) {
    c[i] = a[i] + b[i];
  }
}

int main() {
  int size = 10000;
  int block_size = 128;

  hipError_t error;

  int NO_BYTES = size * sizeof(int);
  // create host pointers
  int *hostA, hostB, hostResults, hostC;

  // allocate memory for host pointers
  hostA = (int *)malloc(NO_BYTES);
  hostB = (int *)malloc(NO_BYTES);
  hostResults = (int *)malloc(NO_BYTES);
  hostC = (int *)malloc(NO_BYTES);

  // init host pointers
  time_t t;

  srand((unsigned)time(&t));

  for (int i = 0; i < size; i++) {
    hostA[i] = (int)(rand() & 0xFF);
  }
  for (int i = 0; i < size; i++) {
    hostB[i] = (int)(rand() & 0xFF);
  }

  // cpu sum array
  clock_t cpu_start, cpu_end;

  cpu_start = clock();
  sum_array_cpu(hostA, hostB, hostC, size);
  cpu_end = clock();





  // device pointers
  int *deviceA, deviceB, deviceC;
  gpuErrchk(hipMalloc((int **)&deviceA, NO_BYTES));
  gpuErrchk(hipMalloc((int **)&deviceB, NO_BYTES));
  gpuErrchk(hipMalloc((int **)&deviceC, NO_BYTES));

  // memory transfer host -> device
  clock_t host_to_device_start, host_to_device_end;
  host_to_device_start = clock();
  hipMemcpy(deviceA, hostA, NO_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, NO_BYTES, hipMemcpyHostToDevice);
  host_to_device_end = clock();

  clock_t gpu_start,gpu_end;

  // kernel launch parameters
  dim3 block(block_size);
  dim3 grid((size / block.x) + 1);

  gpu_start = clock();
  sum_array_gpu<<<grid, block>>>(deviceA, deviceB, deviceC, size);

  // Wait for results
  hipDeviceSynchronize();
  gpu_end = clock();

  // Copy memory back to host
  clock_t device_to_host_start, device_to_host_end;
  device_to_host_start=clock();
  hipMemcpy(hostResults, deviceC, NO_BYTES, hipMemcpyDeviceToHost);
  device_to_host_end = clock();

  // array comparision
  compare_arrays(hostC, hostResults);

  printf("Sum Array CPU Execution Time : %4.6f \n",(double)((double)(cpu_end-cpu_start)/CLOCKS_PER_SECOND));
  printf("Sum Array GPU Execution Time : %4.6f \n",(double)((double)(gpu_end-gpu_start)/CLOCKS_PER_SECOND));
  printf("Host -> Device Memory Transfer Time : %4.6f \n",(double)((double)(host_to_device_end-host_to_device_start)/CLOCKS_PER_SECOND));
  printf("Device -> Host Memory Transfer Time : %4.6f \n",(double)((double)(device_to_host_end-device_to_host_start)/CLOCKS_PER_SECOND));
  printf("Sum Array GPU Total Execution Time : %4.6f \n",(double)((double)(device_to_host_end-host_to_device_start)/CLOCKS_PER_SECOND));
  // free device memory
  hipFree(deviceC);
  hipFree(deviceB);
  hipFree(deviceA);

  // free host memory
  free(hostResults);
  free(hostB);
  free(hostA);

  return EXIT_SUCCESS;
}